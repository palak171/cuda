
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void sumIterative(int *arr, int *result, int n) {
    int idx = threadIdx.x;
    int sum = 0;
    if (idx == 0) {
        for (int i = 0; i < n; ++i) {
            sum += arr[i];
        }
        *result = sum;
    }
}

int main() {
    int n = 1024;
    int *h_arr = new int[n];
    int h_result = 0;

    for (int i = 0; i < n; i++) h_arr[i] = i + 1;

    int *d_arr, *d_result;
    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&d_result, sizeof(int));
    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);

    sumIterative<<<1, 1>>>(d_arr, d_result, n);

    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Iterative Sum: " << h_result << std::endl;

    hipFree(d_arr);
    hipFree(d_result);
    delete[] h_arr;
    return 0;
}
