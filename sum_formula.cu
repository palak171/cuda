
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void sumFormula(int n, int *result) {
    if (threadIdx.x == 0) {
        *result = n * (n + 1) / 2;
    }
}

int main() {
    int n = 1024;
    int h_result = 0;
    int *d_result;

    hipMalloc(&d_result, sizeof(int));
    sumFormula<<<1, 1>>>(n, d_result);
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Formula Sum: " << h_result << std::endl;

    hipFree(d_result);
    return 0;
}
